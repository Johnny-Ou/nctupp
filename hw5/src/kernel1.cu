#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int width, int* img, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;
    int index = thisY * width + thisX;
    
    // img[nidex] = mandel(x, y, maxIterations);
    float tempX = x;
    float tempY = y;
    int i;
    for (i = 0; i < maxIterations; ++i)
    {

        if (tempX * tempX + tempY * tempY > 4.f)
        break;

        float new_tempX = tempX * tempX - tempY * tempY;
        float new_tempY = 2.f * tempX * tempY;
        tempX = x + new_tempX;
        tempY = y + new_tempY;
    }
    
    img[index] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *h_img, *d_img;

    h_img = (int *)malloc(resX * resY * sizeof(int));
    hipMalloc((void **)&d_img, resX * resY * sizeof(int));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlock(resX/threadsPerBlock.x, resY/threadsPerBlock.y);

    mandelKernel<<<numBlock, threadsPerBlock>>>(lowerX, lowerY, stepX, stepY, resX, d_img, maxIterations);
    hipMemcpy(h_img, d_img, resX * resY * sizeof(int), hipMemcpyDeviceToHost);
    memcpy(img, h_img, resX * resY * sizeof(int));

    hipFree(d_img);
    free(h_img);
}