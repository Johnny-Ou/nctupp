#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int width, int* img, int maxIterations, int pitch, int pixelsPerThreads) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    for(int j = 0; j<pixelsPerThreads; j++)
    {
        int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * pixelsPerThreads + j;
        int thisY = blockIdx.y * blockDim.y + threadIdx.y;
        float x = lowerX + thisX * stepX;
        float y = lowerY + thisY * stepY;
        
        // img[nidex] = mandel(x, y, maxIterations);
        float tempX = x;
        float tempY = y;
        int i;
        for (i = 0; i < maxIterations; ++i)
        {

            if (tempX * tempX + tempY * tempY > 4.f)
            break;

            float new_tempX = tempX * tempX - tempY * tempY;
            float new_tempY = 2.f * tempX * tempY;
            tempX = x + new_tempX;
            tempY = y + new_tempY;
        }
        
        int* row = (int *)((char*)img + thisY * pitch);
        row[thisX] = i;
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *h_img, *d_img;
    size_t pitch;
    int pixelsPerThreads = 10; // set number of pixel in group

    hipHostAlloc((void **)&h_img, resX * resY * sizeof(int), hipHostMallocDefault);
    hipMallocPitch((void **)&d_img, &pitch, resX * sizeof(int), resY);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlock(resX/(threadsPerBlock.x * pixelsPerThreads), resY/threadsPerBlock.y);

    mandelKernel<<<numBlock, threadsPerBlock>>>(lowerX, lowerY, stepX, stepY, resX, d_img, maxIterations, pitch, pixelsPerThreads);

    hipMemcpy2D(h_img, resX * sizeof(int), d_img, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, h_img, resX * resY * sizeof(int));

    hipHostFree(h_img);
    hipFree(d_img);
}